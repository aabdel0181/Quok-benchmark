#include <iostream>
#include <random>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

std::default_random_engine generator(2);
std::uniform_real_distribution<float> distribution(0, 1);

#define cudaCheck(err) (cudaErrorCheck(err, __FILE__, __LINE__))
#define cublasCheck(err) (cublasErrorCheck(err, __FILE__, __LINE__))

#define SIZE 2048

void randomize_matrix(float *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = distribution(generator);
    }
}

void const_init_matrix(float *mat, int N, float F)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = F;
    }
}

void cublasErrorCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("[CUDA ERROR] at file %s:%d:\n %s: %s\n", file, line,
               cublasGetStatusName(status), cublasGetStatusString(status));
        exit(EXIT_FAILURE);
    }
}

void cudaErrorCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s: %s\n", file, line,
               hipGetErrorName(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void runCublas(hipblasHandle_t handle, int M, int N, int K, float alpha,
    float *A, float *B, float beta, float *C)
{
    hipblasStatus_t ok = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N, A, K, &beta, C, N);
    cublasCheck(ok);
}

int main() {
    // Setup cublas
    hipblasHandle_t handle;
    cublasCheck(hipblasCreate(&handle));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));

    uint16_t m = SIZE, n = SIZE, k = SIZE;

    // GEMM computes C = α*AB+β*C

    // Just do pure A*B+C (for simpler debugging)
    float alpha = 1.0, beta = 1.0, initC = 1.0;
    float *A = nullptr, *B=nullptr, *C = nullptr;     // host matrices
    float *dA = nullptr, *dB=nullptr, *dC = nullptr; // device matrices

    A = (float *)malloc(sizeof(float) * SIZE * SIZE);
    B = (float *)malloc(sizeof(float) * SIZE * SIZE);
    C = (float *)malloc(sizeof(float) * SIZE * SIZE);

    randomize_matrix(A, SIZE * SIZE);
    randomize_matrix(B, SIZE * SIZE);

    const_init_matrix(C, SIZE * SIZE, initC);

    // A, B, C live in CPU, dA, dB, dC live in GPU
    cudaCheck(hipMalloc((void **)&dA, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dB, sizeof(float) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dC, sizeof(float) * SIZE * SIZE));

    cudaCheck(hipMemcpy(dA, A, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice));

    // Start timing
    cudaCheck(hipEventRecord(start));

    runCublas(handle, m, n, k, alpha, dA, dB, beta, dC);

    // Stop timing
    cudaCheck(hipEventRecord(stop));
    cudaCheck(hipEventSynchronize(stop));

    // Copy result back to host
    hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    cudaCheck(hipEventElapsedTime(&milliseconds, start, stop));

    // Compute GFLOPS (2 * SIZE^3 FLOPs per matrix multiplication)
    double gflops = (2.0 * SIZE * SIZE * SIZE) / (milliseconds * 1e6);
    // Giga Floating Point Operations Per Second
    
    std::cout << "Matrix Size: " << SIZE << "x" << SIZE << std::endl;
    std::cout << "Execution Time: " << milliseconds << " ms" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;

    // Free CPU and GPU memory
    free(A);
    free(B);
    free(C);
    cudaCheck(hipFree(dA));
    cudaCheck(hipFree(dB));
    cudaCheck(hipFree(dC));
    cublasCheck(hipblasDestroy(handle));

    return 0;

}