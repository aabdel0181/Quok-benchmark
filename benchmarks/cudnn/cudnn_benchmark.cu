
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cassert>
#include <cudnn.h>
#include <random>
#include <chrono>
#include <fstream>
#include <iomanip>

// Constants
#define SIZE 256
#define MASK_WIDTH  3 // Note that mask refers to kernel
#define MASK_HEIGHT 3

// Function to pad the input matrix - to ensure that the output has the same size as input
void padInput(const std::vector<float>& input, std::vector<float>& paddedInput,
              int inputWidth, int inputHeight, int kernelWidth, int kernelHeight) {
    int padWidth = kernelWidth / 2;
    int padHeight = kernelHeight / 2;
    int paddedWidth = inputWidth + 2 * padWidth;
    int paddedHeight = inputHeight + 2 * padHeight;

    paddedInput.resize(paddedWidth * paddedHeight, 0.0f);

    for (int y = 0; y < inputHeight; ++y) {
        for (int x = 0; x < inputWidth; ++x) {
            paddedInput[(y + padHeight) * paddedWidth + (x + padWidth)] = input[y * inputWidth + x];
        }
    }
}

// Generate a matrix filled with random values
std::vector<float> generateRandomMatrix(int width, int height) {
    std::vector<float> matrix(width * height);
    std::mt19937 rng(std::chrono::steady_clock::now().time_since_epoch().count());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (auto &val : matrix) val = dist(rng);
    return matrix;
}

// Flip the kernel (for convolution)
void flipKernel(const std::vector<float>& kernel, std::vector<float>& flippedKernel,
                int kernelWidth, int kernelHeight) {
    for (int y = 0; y < kernelHeight; ++y) {
        for (int x = 0; x < kernelWidth; ++x) {
            flippedKernel[y * kernelWidth + x] = kernel[(kernelHeight - 1 - y) * kernelWidth + (kernelWidth - 1 - x)];
        }
    }
}

// Validate convolution with cuDNN
float benchmarkConvolution(const std::vector<float>& input, std::vector<float>& output_cudnn,
                         const std::vector<float>& kernel, int inputWidth, int inputHeight,
                         int kernelWidth, int kernelHeight) {
    int paddedWidth = inputWidth + 2 * (kernelWidth / 2);
    int paddedHeight = inputHeight + 2 * (kernelHeight / 2);
    float *input_device, *kernel_device, *output_device;
    const float alpha = 1.0f, beta = 0.0f;
    const int stride = 1;

    std::vector<float> flippedKernel(kernel.size());
    flipKernel(kernel, flippedKernel, kernelWidth, kernelHeight);
    
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    cudnnTensorDescriptor_t inputDesc, outputDesc;
    cudnnFilterDescriptor_t kernelDesc;
    cudnnConvolutionDescriptor_t convDesc;

    cudnnCreateTensorDescriptor(&inputDesc);
    cudnnSetTensor4dDescriptor(inputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, 1, paddedHeight, paddedWidth);

    cudnnCreateFilterDescriptor(&kernelDesc);
    cudnnSetFilter4dDescriptor(kernelDesc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, 1, 1, kernelHeight, kernelWidth);

    cudnnCreateTensorDescriptor(&outputDesc);
    cudnnSetTensor4dDescriptor(outputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, 1, inputHeight, inputWidth);

    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc, 0, 0, stride, stride, 1, 1, CUDNN_CONVOLUTION, CUDNN_DATA_FLOAT);

    cudnnConvolutionFwdAlgoPerf_t algoPerf;
    int returnedAlgoCount;
    cudnnGetConvolutionForwardAlgorithm_v7(cudnn, inputDesc, kernelDesc, convDesc, outputDesc, 1, &returnedAlgoCount, &algoPerf);
    cudnnConvolutionFwdAlgo_t algo = algoPerf.algo;

    size_t workspaceSize = 0;
    cudnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, kernelDesc, convDesc, outputDesc, algo, &workspaceSize);
    void* workspace = nullptr;
    if (workspaceSize > 0) hipMalloc(&workspace, workspaceSize);
    
    hipMalloc(&input_device, input.size() * sizeof(float));
    hipMalloc(&kernel_device, flippedKernel.size() * sizeof(float));
    hipMalloc(&output_device, output_cudnn.size() * sizeof(float));
    hipMemcpy(input_device, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernel_device, flippedKernel.data(), flippedKernel.size() * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cudnnConvolutionForward(cudnn, &alpha, inputDesc, input_device, kernelDesc, kernel_device,
                            convDesc, algo, workspace, workspaceSize, &beta, outputDesc, output_device);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(output_cudnn.data(), output_device, inputWidth * inputHeight * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(workspace);
    hipFree(input_device);
    hipFree(kernel_device);
    hipFree(output_device);
    cudnnDestroyTensorDescriptor(inputDesc);
    cudnnDestroyTensorDescriptor(outputDesc);
    cudnnDestroyFilterDescriptor(kernelDesc);
    cudnnDestroyConvolutionDescriptor(convDesc);
    cudnnDestroy(cudnn);   
    
    return milliseconds;
}

// Benchmark activation function
float benchmarkActivation(cudnnHandle_t cudnn, cudnnTensorDescriptor_t inputDesc, float* input_device, float* output_device, int inputSize) {
    cudnnActivationDescriptor_t activationDesc;
    cudnnCreateActivationDescriptor(&activationDesc);
    cudnnSetActivationDescriptor(activationDesc, CUDNN_ACTIVATION_RELU, CUDNN_NOT_PROPAGATE_NAN, 0.0);

    const float alpha = 1.0f, beta = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cudnnActivationForward(cudnn, activationDesc, &alpha, inputDesc, input_device, &beta, inputDesc, output_device);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    cudnnDestroyActivationDescriptor(activationDesc);
    return milliseconds;
}

// Benchmark pooling function
float benchmarkPooling(cudnnHandle_t cudnn, cudnnTensorDescriptor_t inputDesc, cudnnTensorDescriptor_t outputDesc, float* input_device, float* output_device) {
    cudnnPoolingDescriptor_t poolingDesc;
    cudnnCreatePoolingDescriptor(&poolingDesc);
    cudnnSetPooling2dDescriptor(poolingDesc, CUDNN_POOLING_MAX, CUDNN_NOT_PROPAGATE_NAN, 2, 2, 0, 0, 2, 2);

    const float alpha = 1.0f, beta = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cudnnPoolingForward(cudnn, poolingDesc, &alpha, inputDesc, input_device, &beta, outputDesc, output_device);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    cudnnDestroyPoolingDescriptor(poolingDesc);
    return milliseconds;
}

int main() {
    const int kernelWidth = MASK_WIDTH;
    const int kernelHeight = MASK_HEIGHT;

    const int inputWidth = SIZE;
    const int inputHeight = SIZE;
    std::vector<float> input = generateRandomMatrix(inputWidth, inputHeight);
    std::vector<float> paddedInput;
    padInput(input, paddedInput, inputWidth, inputHeight, kernelWidth, kernelHeight);
    std::vector<float> kernel(kernelWidth * kernelHeight, 1.0f);
    std::vector<float> output_cudnn(inputWidth * inputHeight, 0);

    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);
    cudnnTensorDescriptor_t inputDesc;
    cudnnCreateTensorDescriptor(&inputDesc);
    cudnnSetTensor4dDescriptor(inputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, 1, inputHeight, inputWidth);
    
    float* input_device;
    float* output_device;
    hipMalloc(&input_device, inputWidth * inputHeight * sizeof(float));
    hipMalloc(&output_device, inputWidth * inputHeight * sizeof(float));
    hipMemcpy(input_device, input.data(), inputWidth * inputHeight * sizeof(float), hipMemcpyHostToDevice);
    
    float cuDNN_time = benchmarkConvolution(paddedInput, output_cudnn, kernel, inputWidth, inputHeight, kernelWidth, kernelHeight);
    float activation_time = benchmarkActivation(cudnn, inputDesc, input_device, output_device, inputWidth * inputHeight);
    float pooling_time = benchmarkPooling(cudnn, inputDesc, inputDesc, input_device, output_device);

    std::cout << "Matrix Size: " << SIZE << "x" << SIZE << "\n";
    std::cout << "Conv Time: " << cuDNN_time << " ms\n";
    std::cout << "Activation Time: " << activation_time << " ms\n";
    std::cout << "Pooling Time: " << pooling_time << " ms\n";
    std::cout << "---------------------------------------------" << std::endl;
    
    hipFree(input_device);
    hipFree(output_device);
    cudnnDestroyTensorDescriptor(inputDesc);
    cudnnDestroy(cudnn);
    return 0;
}
